#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <opencv2\opencv.hpp>
#include "cudarefocus.h"
//#include "cudaRefocus.h"

using namespace std;
using namespace cv;



__device__ int index(int n, int m)
{
	if (0 <= n && n < m)
		return n;
	else if (n < 0)
		return 0;
	else
		return m - 1;
}

__global__ void refocusKernel(uchar* pLFData, uchar* pRefocusLFData, uchar* pRefocusData,float alpha,
                              int imgH, int imgW, int radius)
{
    const int tidx=blockDim.x*blockIdx.x+threadIdx.x;
    const int tidy=blockDim.y*blockIdx.y+threadIdx.y;

    if(tidx == 0 && tidy == 0)
    {
       // printf("KKKKKennel\n");

    }
    int diameter=2*radius+1;
    int heightLF=imgH*diameter;
    int widthLF=imgW*diameter;
    int windowSize=diameter*diameter;
    if (tidx<imgW && tidy<imgH)
    {
        float output_color_B = 0;
		float output_color_G = 0;
		float output_color_R = 0;

		for (int j = -radius; j < radius + 1; j++)
		{
			for (int i = -radius; i < radius + 1; i++)
			{
				float x_ind = i*(1 - 1 / alpha) + tidx;  
				float y_ind = j*(1 - 1 / alpha) + tidy;

                int x_floor = floorf(x_ind);
                int y_floor = floorf(y_ind);

				int x_1 = index(x_floor, imgW); 
				int y_1 = index(y_floor, imgH);
				int x_2 = index(x_floor + 1, imgW); 
				int y_2 = index(y_floor + 1, imgH);

				float x_1_w = 1 - (x_ind - x_floor); 
				float x_2_w = 1 - x_1_w;
				float y_1_w = 1 - (y_ind - y_floor);
				float y_2_w = 1 - y_1_w;

				
				int x_1_index = i + radius + (x_1) * diameter;
				int y_1_index = j + radius + (y_1) * diameter;
				int x_2_index = i + radius + (x_2) * diameter;
				int y_2_index = j + radius + (y_2) * diameter;

				
				float interp_color_B = y_1_w * x_1_w * pLFData[y_1_index * widthLF * 3 + x_1_index * 3 + 0 ] +
									y_2_w * x_1_w * pLFData[y_1_index * widthLF * 3 + x_2_index * 3 + 0 ] +
									y_1_w * x_2_w * pLFData[y_2_index * widthLF * 3 + x_1_index * 3 + 0 ] +
									y_2_w * x_2_w * pLFData[y_2_index * widthLF * 3 + x_2_index * 3 + 0 ];

				float interp_color_G = y_1_w * x_1_w * pLFData[y_1_index * widthLF * 3 + x_1_index * 3 + 1] +
									y_2_w * x_1_w * pLFData[y_1_index * widthLF * 3 + x_2_index * 3 + 1] +
									y_1_w * x_2_w * pLFData[y_2_index * widthLF * 3 + x_1_index * 3 + 1] +
									y_2_w * x_2_w * pLFData[y_2_index * widthLF * 3 + x_2_index * 3 + 1];

				float interp_color_R = y_1_w * x_1_w * pLFData[y_1_index * widthLF * 3 + x_1_index * 3 + 2] +
									y_2_w * x_1_w * pLFData[y_1_index * widthLF * 3 + x_2_index * 3 + 2] +
									y_1_w * x_2_w * pLFData[y_2_index * widthLF * 3 + x_1_index * 3 + 2] +
									y_2_w * x_2_w * pLFData[y_2_index * widthLF * 3 + x_2_index * 3 + 2];

				// CORRESPONDENCE ANALYSIS
				
				int x_index_remap = i + radius + (tidx) * diameter;
				int y_index_remap = j + radius + (tidy) * diameter;
				
				pRefocusLFData[x_index_remap * 3 + y_index_remap * widthLF * 3 + 0] = interp_color_B;
				pRefocusLFData[x_index_remap * 3 + y_index_remap * widthLF * 3 + 1] = interp_color_G;
				pRefocusLFData[x_index_remap * 3 + y_index_remap * widthLF * 3 + 2] = interp_color_R;

				// DEFOCUS ANALYSIS
				output_color_R = interp_color_R + output_color_R;
				output_color_G = interp_color_G + output_color_G;
				output_color_B = interp_color_B + output_color_B;
			}
		}
		
		pRefocusData[tidx * 3 + tidy * imgW * 3 + 0] = output_color_B / windowSize;
		pRefocusData[tidx * 3 + tidy * imgW * 3 + 1] = output_color_G / windowSize;
		pRefocusData[tidx * 3 + tidy * imgW * 3 + 2] = output_color_R / windowSize;
    }
}

void myprint(uint64_t* avail, uint64_t* total )
{

    hipMemGetInfo(avail, total);

}




void cudaRefocus(cv::Mat &LFRemapImg, cv::Mat &refocusImgLF, cv::Mat &refocusImg, float alpha,
                 bool& isFirst, int imgHeightLF, int imgWidthLF, int imgHeight, int imgWidth,
                 int radius, uchar *pLFImg, uchar *pRefocusLFImg, uchar *pRefocusImg,cudaErr * Err)
{
    const char* errStr;

    int sizeLF=imgHeightLF*imgWidthLF*sizeof(uchar3);
    int size=imgHeight*imgWidth*sizeof(uchar3);
    hipError_t err;


    uchar* p1, *p2, *p3;

    hipMalloc((void**)&p1, sizeLF);
    hipMalloc((void**)&p2, sizeLF);
    hipMalloc((void**)&p3, size);

    hipMemcpy(p1, LFRemapImg.data, sizeLF,hipMemcpyHostToDevice);


    dim3 block(8,8);
    dim3 grid( (imgWidth+block.x-1)/block.x, (imgHeight+block.y-1)/block.y);
    refocusKernel<<<grid, block>>>(p1, p2, p3, alpha, imgHeight, imgWidth, radius);
    hipDeviceSynchronize();


    hipMemcpy(refocusImg.data, p3, imgWidth*imgHeight*sizeof(uchar3), hipMemcpyDeviceToHost);

    hipFree(p1);
    hipFree(p2);
    hipFree(p3);

    err=hipGetLastError();
    errStr = hipGetErrorString(err);
    Err->hipError_t = errStr;
    Err->errNum = err;
}

void cudaRefocus_bk(cv::Mat &LFRemapImg, cv::Mat &refocusImgLF, cv::Mat &refocusImg, float alpha,
                 bool& isFirst, int imgHeightLF, int imgWidthLF, int imgHeight, int imgWidth,
                 int radius, uchar *pLFImg, uchar *pRefocusLFImg, uchar *pRefocusImg,cudaErr * Err)
{
    const char* errStr;

    int sizeLF=imgHeightLF*imgWidthLF*sizeof(uchar3);
    int size=imgHeight*imgWidth*sizeof(uchar3);
    hipError_t err;

    if (true==isFirst)
    {
        err=hipMalloc((void**)&pLFImg,sizeLF);
        err=hipMalloc((void**)&pRefocusLFImg, sizeLF);
        err=hipMalloc((void**)&pRefocusImg, size);
        err=hipMemcpy(pLFImg, LFRemapImg.data, sizeLF,hipMemcpyHostToDevice);
        isFirst=false;
    }



    //memcpy
    err = hipMemcpy(pLFImg, LFRemapImg.data, sizeLF, hipMemcpyHostToDevice);


    dim3 block(8,8);
    dim3 grid( (imgWidth+block.x-1)/block.x, (imgHeight+block.y-1)/block.y);
    refocusKernel<<<grid, block>>>(pLFImg, pRefocusLFImg, pRefocusImg,alpha, imgHeight, imgWidth, radius);
    hipDeviceSynchronize();

    if (err!=hipSuccess)
    {
        cout<<" err="<<err<<errStr<<endl;
    }

    err=hipMemcpy(refocusImg.data, pRefocusImg, imgWidth*imgHeight*sizeof(uchar3), hipMemcpyDeviceToHost);

    err=hipGetLastError();
    errStr = hipGetErrorString(err);
    Err->hipError_t = errStr;
    Err->errNum = err;
}
